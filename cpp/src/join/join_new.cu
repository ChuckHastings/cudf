/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/copying.hpp>
#include <cudf/join_new.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/table/table.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/gather.cuh>

#include <join/join_common_utils.hpp>
#include <join/nested_loop_join.cuh>

namespace cudf {

namespace join {

namespace detail {

  // TODO:
  //  sort_merge_join_indices  - Kumar
  //  hash_join_indices        - rework current hash mechanism
  //  filter_join_indices      - new function/kernel shared by sort_merge and hash
  //

  /**
   * @brief Defines nested loop join implementation
   */
  struct nested_loop_join {
    rmm::device_vector<int64_t> operator()(table_view const& left,
                                           table_view const& right,
                                           std::vector<join_operation> const& primary_join_ops,
                                           std::vector<join_operation> const& secondary_join_ops,
                                           hipStream_t stream) {

      return nested_join_indices(left, right, primary_join_ops, stream);
    }
  };

  /**
   * @brief Defines sort merge join implementation
   */
  struct sort_merge_join {
    rmm::device_vector<int64_t> operator()(table_view const& left,
                                           table_view const& right,
                                           std::vector<join_operation> const& primary_join_ops,
                                           std::vector<join_operation> const& secondary_join_ops,
                                           hipStream_t stream) {

      /*
      auto joined_indices = sort_merge_join_indices(left, right, primary_join_ops, stream);

      if (secondary_join_ops.empty()) {
        return joined_indices;
      }

      return filter_join_indices(left, right, joined_indices, secondary_join_ops, stream);
      */
      return rmm::device_vector<int64_t>{};
    }
  };

  /**
   * @brief Defines hash join implementation
   */
  struct hash_join {
    rmm::device_vector<int64_t> operator()(table_view const& left,
                                           table_view const& right,
                                           std::vector<join_operation> const& primary_join_ops,
                                           std::vector<join_operation> const& secondary_join_ops,
                                           hipStream_t stream) {

      /*
      auto joined_indices = hash_join_indices(left, right, primary_join_ops, stream);

      if (secondary_join_ops.empty()) {
        return joined_indices;
      }

      return filter_join_indices(left, right, joined_indices, secondary_join_ops, stream);
      */
      return rmm::device_vector<int64_t>{};
    }
  };
    
  /**
   * @brief determine if the specified join is a trivial join resulting in
   * an empty table.
   *
   * Checks for simple edge conditions that can short-circuit the expensive join
   * computations (e.g. empty inputs).
   *
   * @param[in] left                The left table
   * @param[in] right               The right table
   * @param[in] primary_join_ops    Vector of join operations used as the primary join
   * @param[in] secondary_join_ops  Vector of join operations used as the secondary join
   * @param[in] JoinKind            Type of join (INNER_JOIN, LEFT_JOIN, FULL_JOIN)
   *
   * @return true if the result is trivial, false if it needs to be computed
   */
  bool is_trivial_join(table_view const& left,
                       table_view const& right,
                       std::vector<join_operation> const& primary_join_ops,
                       std::vector<join_operation> const& secondary_join_ops,
                       join_kind JoinKind) {

    // If there is nothing to join, then send empty table with all columns
    if (primary_join_ops.empty() && secondary_join_ops.empty()) {
      return true;
    }

    // If the inputs are empty, immediately return
    if ((0 == left.num_rows()) && (0 == right.num_rows())) {
      return true;
    }

    // If left join and the left table is empty, return immediately
    if ((join_kind::LEFT_JOIN == JoinKind) && (0 == left.num_rows())) {
      return true;
    }

    // If Inner Join and either table is empty, return immediately
    if ((join_kind::INNER_JOIN == JoinKind) &&
        ((0 == left.num_rows()) || (0 == right.num_rows()))) {
      return true;
    }

    return false;
  }

  /**
   * @brief Returns a vector with non-common indices which is set difference
   * between `[0, num_columns)` and index values in common_column_indices
   *
   * @param num_columns           The number of columns, which represents column
   *                              indices from `[0, num_columns)` in a table
   * @param common_column_indices Vector of common indices which needs to be
   *                              excluded from `[0, num_columns)`
   * @return vector               Vector containing only the indices which are
   *                              not present in `common_column_indices`
   */
  auto non_common_column_indices(size_type num_columns,
                                 std::vector<size_type> const& common_column_indices) {
    CUDF_EXPECTS(common_column_indices.size() <= static_cast<unsigned long>(num_columns),
                 "Too many columns in common");
    std::vector<size_type> all_column_indices(num_columns);
    std::iota(std::begin(all_column_indices), std::end(all_column_indices), 0);
    std::vector<size_type> sorted_common_column_indices{common_column_indices};
    std::sort(std::begin(sorted_common_column_indices), std::end(sorted_common_column_indices));
    std::vector<size_type> non_common_column_indices(num_columns - common_column_indices.size());
    std::set_difference(std::cbegin(all_column_indices),
                        std::cend(all_column_indices),
                        std::cbegin(sorted_common_column_indices),
                        std::cend(sorted_common_column_indices), std::begin(non_common_column_indices));
    return non_common_column_indices;
  }

  /**
   * @brief  Construct an empty joined table of the proper structure.
   *
   * @param[in] left               The left table
   * @param[in] right              The right table
   * @param[in] columns_in_common  List of columns in common, only return one of these.
   *
   * @return table with proper structure but no rows.
   */
  std::unique_ptr<experimental::table> get_empty_joined_table(table_view const& left,
                                                              table_view const& right,
                                                              std::vector<std::pair<size_type, size_type>> const& columns_in_common) {
    std::vector<size_type> right_columns_in_common (columns_in_common.size());
    std::transform(columns_in_common.begin(),
                   columns_in_common.end(),
                   right_columns_in_common.begin(),
                   [](auto& col) { return col.second; } );
    std::unique_ptr<experimental::table> empty_left = experimental::empty_like(left);
    std::unique_ptr<experimental::table> empty_right = experimental::empty_like(right);
    std::vector <size_type> right_non_common_indices = non_common_column_indices(right.num_columns(), right_columns_in_common);
    table_view tmp_right_table = (*empty_right).select(right_non_common_indices);
    table_view tmp_table{{*empty_left, tmp_right_table}};
    return std::make_unique<experimental::table>(tmp_table);
  }
  
  /**
   * @brief  Compute a list of indices that are not referenced in the join intermediate
   *         output.  This will be used to populate portions of LEFT and FULL join
   *         results.
   *
   *  The intermediate output of the joins is an INNER_JOIN.  In order to convert to
   *  a LEFT_JOIN we need to identify which rows in the left table are not referenced.
   *  In order to convert to a FULL_JOIN we need to identify which rows in the right
   *  table are not referenced.
   *
   *  @tparam iterator      The type of the iterator
   *
   *  @param indices_begin   Iterator pointing to beginning of the used indices
   *  @param join_size       Number of elements in the collection we're iterating over
   *  @param row_count       Number of elements in the table we want to complement
   *
   *  @return                Device vector containing all row indices that are not
   *                         referenced in the provided iterator.
   */
  template <typename iterator>
  rmm::device_vector<cudf::size_type> get_indices_complement(iterator indices_begin,
                                                             cudf::size_type join_size,
                                                             cudf::size_type row_count,
                                                             hipStream_t stream) {

    //Get array of indices that do not appear in indices

    //Vector allocated for unmatched result
    rmm::device_vector<cudf::size_type> indices_complement(row_count);

    //
    // NOTE:  In this implementation, the indices_begin and indices_end
    //        iterators point to a range of elements from an INNER_JOIN,
    //        meaning they only include valid values within the table
    //
    rmm::device_vector<cudf::size_type> invalid_index_map(row_count, 1);

    //invalid_index_map[index_ptr[i]] = 0 for i = 0 to row_count
    //Thus specifying that those locations are valid
    thrust::scatter(rmm::exec_policy(stream)->on(stream),
                    thrust::make_constant_iterator(0),
                    thrust::make_constant_iterator(0) + join_size,
                    indices_begin,                      //Index locations
                    invalid_index_map.begin());         //Output indices

    //Create list of indices that have been marked as invalid
    auto copy_end = thrust::copy_if(rmm::exec_policy(stream)->on(stream),
                                    thrust::make_counting_iterator<cudf::size_type>(0),
                                    thrust::make_counting_iterator<cudf::size_type>(row_count),
                                    invalid_index_map.begin(),
                                    indices_complement.begin(),
                                    thrust::identity<size_type>());

    cudf::size_type indices_count = thrust::distance(indices_complement.begin(), copy_end);

    indices_complement.resize(indices_count);
    return indices_complement;
  }

  /**
   * @brief  Combines the non common left, common left and non common right
   *         columns in the correct order to form the join output table.
   *
   * @param[in] left_noncommon_cols        Columns obtained by gathering non common left
   *                                       columns.
   * @param[in] left_noncommon_col_indices Output locations of non common left columns
   *                                       in the final table output
   * @param[in] left_common_cols           Columns obtained by gathering common left
   *                                       columns.
   * @param[in] left_common_col_indices    Output locations of common left columns in the
   *                                       final table output
   * @param[in] right_noncommon_cols       Table obtained by gathering non common right
   *                                       columns.
   *
   * @return  table containing rearranged columns.
   */
  std::vector<std::unique_ptr<column>> combine_join_columns(std::vector<std::unique_ptr<column>>&& left_noncommon_cols,
                                                            std::vector<size_type> const& left_noncommon_col_indices,
                                                            std::vector<std::unique_ptr<column>>&& left_common_cols,
                                                            std::vector<size_type> const& left_common_col_indices,
                                                            std::vector<std::unique_ptr<column>>&& right_noncommon_cols) {

    std::vector<std::unique_ptr<column>> combined_cols(left_noncommon_cols.size() + left_common_cols.size());
    for(size_t i = 0; i < left_noncommon_cols.size(); ++i) {
      combined_cols.at(left_noncommon_col_indices.at(i)) = std::move(left_noncommon_cols.at(i));
    }
    for(size_t i = 0; i < left_common_cols.size(); ++i) {
      combined_cols.at(left_common_col_indices.at(i)) = std::move(left_common_cols.at(i));
    }
    combined_cols.insert(combined_cols.end(),
                         std::make_move_iterator(right_noncommon_cols.begin()),
                         std::make_move_iterator(right_noncommon_cols.end()));
    return combined_cols;
  }

  /**
   * @brief Iterator that allows us to convert a device vector of int64_t
   *        generated by the join functions into left and right row offsets.
   *
   * This iterator can operate on just the inner join, or it can operate on
   * the inner join and the appropriate complement sets.  It will return the
   * JoinNoneValue when in the left and right complement sets as appropriate.
   *
   * The order of rows is as follows:
   *
   *    1) The first range of elements is from the inner join.  This is always
   *       present.
   *    2) The second range of elements is from left_complement.  If we're doing
   *       a left join or a full join, this range will be specified.  Any
   *       reference in this range of elements will return the left offset
   *       from the left_complement vector or it will return JoinNoneValue if
   *       we want the right offset.
   *    3) The third range of elements is from right_complement.  If we're
   *       doing a full join, this range will be specified.  Any reference in
   *       this range will return the right offset from the right complement vector
   *       or it will return JoinNoneValue if we want the left offset.
   *
   * Note that left_complement and right_complement can be empty either because we
   * don't want them or because there are no values.  This class doesn't care.
   *
   * @tparam  left_index     true if we want the left row offset,
   *                         false if we want the right row offset
   *
   */
  template <bool left_index>
  struct join_output_iterator {
    /**
     *  @brief   Host-side constructor
     *
     *  @param joined_indices    Vector of joined indices.  Each index is encoded as:
     *                           (left_offset * right_num_rows) + right_offset
     *  @param left_complement   Vector of offsets in left that are not referenced
     *                           in the inner join.  Should be empty if we want
     *                           an inner join.
     *  @param right_complement  Vector of offsets in right that are not referenced
     *                           in the inner join.  Should be empty unless we want
     *                           a full join.
     *  @param right_num_rows    The number of rows in the right table (used to
     *                           decode the joined index).
     */
    __host__ join_output_iterator(rmm::device_vector<int64_t> &joined_indices,
                                  rmm::device_vector<cudf::size_type> &left_complement,
                                  rmm::device_vector<cudf::size_type> &right_complement,
                                  cudf::size_type right_num_rows):
      _joined_indices(joined_indices.data().get()),
      _left_complement(left_complement.data().get()),
      _right_complement(right_complement.data().get()),
      _joined_indices_size(joined_indices.size()),
      _left_complement_size(left_complement.size()),
      _right_complement_size(right_complement.size()),
      _right_num_rows(right_num_rows)  {}
    
    /**
     *  @brief Device-side operator to return the proper result.
     */
    __device__ cudf::size_type operator()(cudf::size_type index) {
      if (index < _joined_indices_size) {
        if (left_index) {
          return _joined_indices[index] / _right_num_rows;
        } else {
          return _joined_indices[index] % _right_num_rows;
        }
      }

      index -= _joined_indices_size;
      if (index < _left_complement_size) {
        if (left_index) {
          return _left_complement[index];
        } else {
          return JoinNoneValue;
        }
      }

      index -= _left_complement_size;
      if (left_index) {
        return JoinNoneValue;
      } else {
        return _right_complement[index];
      }
    }

  private:
    int64_t          *_joined_indices;
    cudf::size_type  *_left_complement;
    cudf::size_type  *_right_complement;
    cudf::size_type   _joined_indices_size;
    cudf::size_type   _left_complement_size;
    cudf::size_type   _right_complement_size;
    cudf::size_type   _right_num_rows;
  };
  
  /**
   * @brief  Gathers rows from `left` and `right` table and combines them into a
   * single table.
   *
   * @param left Left input table
   * @param right Right input table
   * @param joined_indices Pair of vectors containing row indices from which
   * `left` and `right` tables are gathered. If any row index is out of bounds,
   * the contribution in the output `table` will be NULL.
   * @param columns_in_common is a vector of pairs of column indices
   * from tables `left` and `right` respectively, that are "in common".
   * For "common" columns, only a single output column will be produced.
   * For an inner or left join, the result will be gathered from the column in
   * `left`. For a full join, the result will be gathered from both common
   * columns in `left` and `right` and concatenated to form a single column.
   *
   * @Returns `table` containing the concatenation of rows from `left` and
   * `right` specified by `joined_indices`.
   * For any columns indicated by `columns_in_common`, only the corresponding
   * column in `left` will be included in the result. Final form would look like
   * `left(including common columns)+right(excluding common columns)`.
   */
  template <join_kind JoinKind>
  std::unique_ptr<experimental::table> construct_join_output_df(table_view const& left,
                                                                table_view const& right,
                                                                rmm::device_vector<int64_t> & joined_indices,
                                                                std::vector<std::pair<size_type, size_type>> const& columns_in_common,
                                                                rmm::mr::device_memory_resource* mr,
                                                                hipStream_t stream) {

    if ((join_kind::INNER_JOIN == JoinKind) && (joined_indices.size() == 0)) {
      return get_empty_joined_table(left, right, columns_in_common);
    }
    
    std::vector<size_type> left_common_col;
    left_common_col.reserve(columns_in_common.size());
    std::vector<size_type> right_common_col;
    right_common_col.reserve(columns_in_common.size());
    for (const auto c : columns_in_common) {
      left_common_col.push_back(c.first);
      right_common_col.push_back(c.second);
    }
    std::vector<size_type> left_noncommon_col = non_common_column_indices(left.num_columns(), left_common_col);
    std::vector<size_type> right_noncommon_col = non_common_column_indices(right.num_columns(), right_common_col);

    //
    //   inner_join_indices only gives us part of the solution if we want
    //   left join or right join.  So we need to get what we need for the output
    //   dataframe.
    //
    rmm::device_vector<cudf::size_type> left_indices_complement{};
    rmm::device_vector<cudf::size_type> right_indices_complement{};

    join_output_iterator<true>  left_iterator(joined_indices, left_indices_complement, right_indices_complement, right.num_rows());
    join_output_iterator<false> right_iterator(joined_indices, left_indices_complement, right_indices_complement, right.num_rows());
    
    cudf::size_type output_size = joined_indices.size();

    bool const nullify_out_of_bounds{ JoinKind != join_kind::INNER_JOIN };

    if (join_kind::LEFT_JOIN == JoinKind) {
      left_indices_complement = get_indices_complement(thrust::make_transform_iterator(thrust::make_counting_iterator<cudf::size_type>(0), left_iterator),
                                                       output_size,
                                                       left.num_rows(),
                                                       stream);
      output_size += left_indices_complement.size();
    }

    if (join_kind::FULL_JOIN == JoinKind) {
      left_indices_complement = get_indices_complement(thrust::make_transform_iterator(thrust::make_counting_iterator<cudf::size_type>(0), left_iterator),
                                                       output_size,
                                                       left.num_rows(),
                                                       stream);
      right_indices_complement = get_indices_complement(thrust::make_transform_iterator(thrust::make_counting_iterator<cudf::size_type>(0), right_iterator),
                                                        output_size,
                                                        right.num_rows(),
                                                        stream);

      output_size += left_indices_complement.size() + right_indices_complement.size();
    }

    std::unique_ptr<experimental::table> common_table;

    //
    //  Update in case they changed above
    //
    left_iterator = join_output_iterator<true>(joined_indices, left_indices_complement, right_indices_complement, right.num_rows());
    right_iterator = join_output_iterator<false>(joined_indices, left_indices_complement, right_indices_complement, right.num_rows());

    // Construct the joined columns
    if (not columns_in_common.empty()) {
      if (join_kind::FULL_JOIN == JoinKind) {
        auto common_from_right = experimental::detail::gather(right.select(right_common_col),
                                                              right_indices_complement.begin(),
                                                              right_indices_complement.end(),
                                                              false, nullify_out_of_bounds);
        auto common_from_left = experimental::detail::gather(left.select(left_common_col),
                                                             thrust::make_transform_iterator(thrust::make_counting_iterator<cudf::size_type>(0), left_iterator),
                                                             thrust::make_transform_iterator(thrust::make_counting_iterator<cudf::size_type>(joined_indices.size() + left_indices_complement.size()), left_iterator),
                                                             false, nullify_out_of_bounds);

        common_table = experimental::concatenate({common_from_left->view(), common_from_right->view()});
      } else {
        common_table = experimental::detail::gather(left.select(left_common_col),
                                                    thrust::make_transform_iterator(thrust::make_counting_iterator<cudf::size_type>(0), left_iterator),
                                                    thrust::make_transform_iterator(thrust::make_counting_iterator<cudf::size_type>(output_size), left_iterator),
                                                    false, nullify_out_of_bounds);
      }
    }

    // Construct the left non common columns
    auto left_table = experimental::detail::gather(left.select(left_noncommon_col),
                                                   thrust::make_transform_iterator(thrust::make_counting_iterator<cudf::size_type>(0), left_iterator),
                                                   thrust::make_transform_iterator(thrust::make_counting_iterator<cudf::size_type>(output_size), left_iterator),
                                                   false, nullify_out_of_bounds);

    auto right_table = experimental::detail::gather(right.select(right_noncommon_col),
                                                    thrust::make_transform_iterator(thrust::make_counting_iterator<cudf::size_type>(0), right_iterator),
                                                    thrust::make_transform_iterator(thrust::make_counting_iterator<cudf::size_type>(output_size), right_iterator),
                                                    false, nullify_out_of_bounds);

    return std::make_unique<experimental::table>(combine_join_columns(left_table->release(), left_noncommon_col,
                                                                      common_table->release(), left_common_col,
                                                                      right_table->release()));
  }
  
  /**
   *  @brief  Core join implementation.
   *
   *  Provides the basic structure of the join implementation for INNER_JOIN, LEFT_JOIN and FULL_JOIN.
   *
   *  @tparam JoinKind            The type of join (INNER_JOIN, LEFT_JOIN, FULL_JOIN)
   *  @tparam join_indices_type   A class defining a functor implementing the desired join
   *
   * @param[in] left               The left table
   * @param[in] right              The right table
   * @param[in] primary_join_ops   The primary join operations.  Each join operation identifies a
   *                               comparison operator and a pair of columns.  The join operations in the
   *                               vector are logically combined with an AND.  The primary_join_ops
   *                               is used as the sort/merge key for creating an intermediate result.
   * @param[in] secondary_join_ops The secondary join operations.  Each join operation identifies a
   *                               comparison operator and a pair of columns.  The join operations in the
   *                               vector are logically combined with an AND.  The secondary_join_ops
   *                               is used as a filter on the intermediate result to create the final result.
   * @param[in] columns_in_common  A vector of pairs of column indices into `left` and `right`,
   *                               respectively, that are "in common". For "common" columns, only a
   *                               single output column will be produced, which is gathered from `left_on`
   *                               columns.  Columns from left and right which are not identified within
   *                               columns_in_common will also be output.
   * @param[in] join_indices_impl  Object defining the implementation of the desired join.
   * @param[in] mr                 Memory resource used to allocate the returned table and columns
   * @param[in] stream             Cuda stream
   *
   * @returns                      Result of joining `left` and `right` tables on the columns
   *                               specified by join_ops.  The resulting table will be joined columns of
   *                               `left(common columns)+left(excluding common columns)+right(excluding common columns)`.
   */
  template <join_kind JoinKind, typename join_indices_type>
  std::unique_ptr<experimental::table> join(cudf::table_view const& left,
                                            cudf::table_view const& right,
                                            std::vector<join_operation> const& primary_join_ops,
                                            std::vector<join_operation> const& secondary_join_ops,
                                            std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                            join_indices_type join_indices_impl,
                                            rmm::mr::device_memory_resource* mr,
                                            hipStream_t stream = 0) {

    CUDF_EXPECTS (0 != left.num_columns(), "Left table is empty");
    CUDF_EXPECTS (0 != right.num_columns(), "Right table is empty");

    for (auto p : columns_in_common) {
      CUDF_EXPECTS(left.column(p.first).type() == right.column(p.second).type(), "Mismatch in columns in common data types");
    }

    for (auto p : primary_join_ops) {
      CUDF_EXPECTS(left.column(p.left_column_idx).type() == right.column(p.right_column_idx).type(), "Mismatch in primary_join_ops data types");
    }

    for (auto p : secondary_join_ops) {
      CUDF_EXPECTS(left.column(p.left_column_idx).type() == right.column(p.right_column_idx).type(), "Mismatch in secondary_join_ops data types");
    }

    if (is_trivial_join(left, right, primary_join_ops, secondary_join_ops, JoinKind)) {
      return get_empty_joined_table(left, right, columns_in_common);
    }

    if ((join_kind::FULL_JOIN == JoinKind) && (0 == left.num_rows())) {
      //
      //  write this example.  Return a table with nulls for left columns and all of the
      //  right columns
      //
    }

    //
    //  Call the specific join implementation.  Note that in the new implementation join_indices_impl
    //  only computes the INNER_JOIN.  If a LEFT_JOIN or FULL_JOIN is desired that is addressed
    //  in construction the output data frame below.
    //
    auto joined_indices = join_indices_impl(left, right, primary_join_ops, secondary_join_ops, stream);

    return construct_join_output_df<JoinKind>(left, right, joined_indices, columns_in_common, mr, stream);
  }
} //namespace detail

namespace inner_join {
  std::unique_ptr<experimental::table> nested_loop(cudf::table_view const& left,
                                                   cudf::table_view const& right,
                                                   std::vector<join_operation> const& join_ops,
                                                   std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                                   rmm::mr::device_memory_resource* mr) {

    return detail::join<detail::join_kind::INNER_JOIN>(left, right, join_ops, join_ops, columns_in_common, detail::nested_loop_join{}, mr);
  }

  std::unique_ptr<experimental::table> sort_merge(cudf::table_view const& left,
                                                  cudf::table_view const& right,
                                                  std::vector<join_operation> const& primary_join_ops,
                                                  std::vector<join_operation> const& secondary_join_ops,
                                                  std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                                  rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::INNER_JOIN>(left, right, primary_join_ops, secondary_join_ops, columns_in_common, detail::sort_merge_join{}, mr);
  }

  std::unique_ptr<experimental::table> hash(cudf::table_view const& left,
                                            cudf::table_view const& right,
                                            std::vector<join_operation> const& primary_join_ops,
                                            std::vector<join_operation> const& secondary_join_ops,
                                            std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                            rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::INNER_JOIN>(left, right, primary_join_ops, secondary_join_ops, columns_in_common, detail::hash_join{}, mr);
  }
} //namespace inner_join

namespace left_join {
  std::unique_ptr<experimental::table> nested_loop(cudf::table_view const& left,
                                                   cudf::table_view const& right,
                                                   std::vector<join_operation> const& join_ops,
                                                   std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                                   rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::LEFT_JOIN>(left, right, join_ops, join_ops, columns_in_common, detail::nested_loop_join{}, mr);
  }

  std::unique_ptr<experimental::table> sort_merge(cudf::table_view const& left,
                                                  cudf::table_view const& right,
                                                  std::vector<join_operation> const& primary_join_ops,
                                                  std::vector<join_operation> const& secondary_join_ops,
                                                  std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                                  rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::LEFT_JOIN>(left, right, primary_join_ops, secondary_join_ops, columns_in_common, detail::sort_merge_join{}, mr);
  }

  std::unique_ptr<experimental::table> hash(cudf::table_view const& left,
                                            cudf::table_view const& right,
                                            std::vector<join_operation> const& primary_join_ops,
                                            std::vector<join_operation> const& secondary_join_ops,
                                            std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                            rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::LEFT_JOIN>(left, right, primary_join_ops, secondary_join_ops, columns_in_common, detail::hash_join{}, mr);
  }

} //namespace left_join

namespace full_join {
  std::unique_ptr<experimental::table> nested_loop(cudf::table_view const& left,
                                                   cudf::table_view const& right,
                                                   std::vector<join_operation> const& join_ops,
                                                   std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                                   rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::FULL_JOIN>(left, right, join_ops, join_ops, columns_in_common, detail::nested_loop_join{}, mr);
  }

  std::unique_ptr<experimental::table> sort_merge(cudf::table_view const& left,
                                                  cudf::table_view const& right,
                                                  std::vector<join_operation> const& primary_join_ops,
                                                  std::vector<join_operation> const& secondary_join_ops,
                                                  std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                                  rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::FULL_JOIN>(left, right, primary_join_ops, secondary_join_ops, columns_in_common, detail::sort_merge_join{}, mr);
  }

  std::unique_ptr<experimental::table> hash(cudf::table_view const& left,
                                            cudf::table_view const& right,
                                            std::vector<join_operation> const& primary_join_ops,
                                            std::vector<join_operation> const& secondary_join_ops,
                                            std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                            rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::FULL_JOIN>(left, right, primary_join_ops, secondary_join_ops, columns_in_common, detail::hash_join{}, mr);
  }
} //namespace full_join

namespace nested_loop {
  std::unique_ptr<experimental::table> inner_join(cudf::table_view const& left,
                                                  cudf::table_view const& right,
                                                  std::vector<join_operation> const& join_ops,
                                                  std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                                  rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::INNER_JOIN>(left, right, join_ops, join_ops, columns_in_common, detail::nested_loop_join{}, mr);
  }

  std::unique_ptr<experimental::table> left_join(cudf::table_view const& left,
                                                 cudf::table_view const& right,
                                                 std::vector<join_operation> const& join_ops,
                                                 std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                                 rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::LEFT_JOIN>(left, right, join_ops, join_ops, columns_in_common, detail::nested_loop_join{}, mr);
  }

  std::unique_ptr<experimental::table> full_join(cudf::table_view const& left,
                                                 cudf::table_view const& right,
                                                 std::vector<join_operation> const& join_ops,
                                                 std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                                 rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::FULL_JOIN>(left, right, join_ops, join_ops, columns_in_common, detail::nested_loop_join{}, mr);
  }
} //namespace nested_loop

namespace sort_merge {
  std::unique_ptr<experimental::table> inner_join(cudf::table_view const& left,
                                                  cudf::table_view const& right,
                                                  std::vector<join_operation> const& primary_join_ops,
                                                  std::vector<join_operation> const& secondary_join_ops,
                                                  std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                                  rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::INNER_JOIN>(left, right, primary_join_ops, secondary_join_ops, columns_in_common, detail::sort_merge_join{}, mr);
  }

  std::unique_ptr<experimental::table> left_join(cudf::table_view const& left,
                                                 cudf::table_view const& right,
                                                 std::vector<join_operation> const& primary_join_ops,
                                                 std::vector<join_operation> const& secondary_join_ops,
                                                 std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                                 rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::LEFT_JOIN>(left, right, primary_join_ops, secondary_join_ops, columns_in_common, detail::sort_merge_join{}, mr);
  }

  std::unique_ptr<experimental::table> full_join(cudf::table_view const& left,
                                                 cudf::table_view const& right,
                                                 std::vector<join_operation> const& primary_join_ops,
                                                 std::vector<join_operation> const& secondary_join_ops,
                                                 std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                                 rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::FULL_JOIN>(left, right, primary_join_ops, secondary_join_ops, columns_in_common, detail::sort_merge_join{}, mr);
  }
} //namespace sort_merge
  
namespace hash {
  std::unique_ptr<experimental::table> inner_join(cudf::table_view const& left,
                                                  cudf::table_view const& right,
                                                  std::vector<join_operation> const& primary_join_ops,
                                                  std::vector<join_operation> const& secondary_join_ops,
                                                  std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                                  rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::INNER_JOIN>(left, right, primary_join_ops, secondary_join_ops, columns_in_common, detail::hash_join{}, mr);
  }

  std::unique_ptr<experimental::table> left_join(cudf::table_view const& left,
                                                 cudf::table_view const& right,
                                                 std::vector<join_operation> const& primary_join_ops,
                                                 std::vector<join_operation> const& secondary_join_ops,
                                                 std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                                 rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::LEFT_JOIN>(left, right, primary_join_ops, secondary_join_ops, columns_in_common, detail::hash_join{}, mr);
  }

  std::unique_ptr<experimental::table> full_join(cudf::table_view const& left,
                                                 cudf::table_view const& right,
                                                 std::vector<join_operation> const& primary_join_ops,
                                                 std::vector<join_operation> const& secondary_join_ops,
                                                 std::vector<std::pair<cudf::size_type, cudf::size_type>> const& columns_in_common,
                                                 rmm::mr::device_memory_resource* mr) {
    return detail::join<detail::join_kind::FULL_JOIN>(left, right, primary_join_ops, secondary_join_ops, columns_in_common, detail::hash_join{}, mr);
  }
} //namespace hash
  
} //namespace join

} //namespace cudf
